#include "hip/hip_runtime.h"
#include <stddef.h>
#include <unitypes.h>
#include "../main/constants.h"
#include <stdlib.h>
#include <stdio.h>
#include "model_gpu_utils.h"

#define ENDO
#define NEQ 12


static __device__ size_t pitch;
static size_t pitch_h;

__global__ void kernel_set_model_inital_conditions(Real *sv, int num_volumes);

__global__ void solve_gpu(Real dt, Real *sv, Real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps, Real *fibrosis, Real atpi);

inline __device__ void RHS_gpu(Real *sv_, Real *rDY_, Real stim_current, int threadID_, Real dt, Real fibrosis, Real atpi);


extern "C" size_t set_model_initial_conditions_gpu(Real **sv, uint32_t num_volumes) {

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(Real);

    check_cuda_error(hipMallocPitch((void **) &(*sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));


    kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(*sv, num_volumes);

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();
    return pitch_h;

}


extern "C" void solve_model_odes_gpu(Real dt, Real *sv, Real *stim_currents, uint32_t *cells_to_solve,
                                    uint32_t num_cells_to_solve, int num_steps, void *extra_data,
                                    size_t extra_data_bytes_size) {


    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t stim_currents_size = sizeof(Real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    Real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    Real atpi = 6.8;
    Real *fibrosis_device;
    Real *fibs = NULL;

    if(extra_data) {
        atpi = ((Real*)extra_data)[0];
        fibs = ((Real*)extra_data)+1;
    }
    else {
        fibs = (Real*)calloc(num_cells_to_solve, sizeof(Real));
    }

    check_cuda_error(hipMalloc((void **) &fibrosis_device, extra_data_bytes_size-sizeof(Real)));
    check_cuda_error(hipMemcpy(fibrosis_device, fibs, extra_data_bytes_size-sizeof(Real), hipMemcpyHostToDevice));

    solve_gpu<<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps, fibrosis_device, atpi);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    check_cuda_error(hipFree(fibrosis_device));

    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if(!extra_data) free(fibs);
}


__global__ void kernel_set_model_inital_conditions(Real *sv, int num_volumes)
{
    // Thread ID
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if(threadID < num_volumes) {

        *((Real *) ((char *) sv + pitch * 0) + threadID) = -86.2f;   // V;       millivolt
        *((Real *) ((char *) sv + pitch * 1) + threadID) = 0.0f; //M
        *((Real *) ((char *) sv + pitch * 2) + threadID) = 0.75; //H
        *((Real *) ((char *) sv + pitch * 3) + threadID) = 0.75; //J
        *((Real *) ((char *) sv + pitch * 4) + threadID) = 0.0f; //Xr1
        *((Real *) ((char *) sv + pitch * 5) + threadID) = 0.0f; //Xs
        *((Real *) ((char *) sv + pitch * 6) + threadID) = 1.0; //S
        *((Real *) ((char *) sv + pitch * 7) + threadID) = 1.0; //F
        *((Real *) ((char *) sv + pitch * 8) + threadID) = 1.0; //F2
        *((Real *) ((char *) sv + pitch * 9) + threadID) = 0.0; //D_INF
        *((Real *) ((char *) sv + pitch * 10) + threadID) = 0.0; //R_INF
        *((Real *) ((char *) sv + pitch * 11) + threadID) = 0.0; //Xr2_INF
    }
}


// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(Real dt, Real *sv, Real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps, Real *fibrosis,  Real atpi)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        Real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            RHS_gpu(sv, rDY, stim_currents[threadID], sv_id, dt, fibrosis[threadID], atpi);

            *((Real*)((char*)sv) + sv_id) = dt*rDY[0] + *((Real*)((char*)sv) + sv_id);

            for(int i = 1; i < 12; i++) {
                *((Real*)((char*)sv + pitch * i) + sv_id) = rDY[i];
            }

        }

    }
}


inline __device__ void RHS_gpu(Real *sv_, Real *rDY_, Real stim_current, int threadID_, Real dt, Real fibrosis, Real atpi) {

    const Real svolt = *((Real*)((char*)sv_ + pitch * 0) + threadID_);

    const Real sm   = *((Real*)((char*)sv_ + pitch * 1) + threadID_);
    const Real sh   = *((Real*)((char*)sv_ + pitch * 2) + threadID_);
    const Real sj   = *((Real*)((char*)sv_ + pitch * 3) + threadID_);
    const Real sxr1 = *((Real*)((char*)sv_ + pitch * 4) + threadID_);
    const Real sxs  = *((Real*)((char*)sv_ + pitch * 5) + threadID_);
    const Real ss   = *((Real*)((char*)sv_ + pitch * 6) + threadID_);
    const Real  sf  = *((Real*)((char*)sv_ + pitch * 7) + threadID_);
    const Real sf2  = *((Real*)((char*)sv_ + pitch * 8) + threadID_);
    const Real D_INF  = *((Real*)((char*)sv_ + pitch * 9) + threadID_);
    const Real Xr2_INF  = *((Real*)((char*)sv_ + pitch * 10) + threadID_);
    const Real R_INF  = *((Real*)((char*)sv_ + pitch * 11) + threadID_);

    const Real natp = 0.24;          // K dependence of ATP-sensitive K current
    const Real nicholsarea = 0.00005; // Nichol's areas (cm^2)
    const Real hatp = 2;             // Hill coefficient

    Real Ko   = 5.4;

    Real atpi_change = 6.8f-atpi;

    atpi = atpi + atpi_change*fibrosis;

    //Real katp = 0.306;
    const Real katp = -0.0942857142857f*atpi + 0.683142857143f; //Ref: A Comparison of Two Models of Human Ventricular Tissue: Simulated Ischaemia and Re-entry    


    const Real patp =  1.0f/(1.0f + powf((atpi/katp),hatp));
    const Real gkatp    =  0.000195f/nicholsarea;
    const Real gkbaratp =  gkatp*patp*powf((Ko/4),natp);

    const Real katp2= 1.4;
    const Real hatp2 = 2.6;
    const Real pcal = 1.0f/(1.0f + powf((katp2/atpi),hatp2));

    const Real Cao=2.0;
    const Real Nao=140.0;
    const Real Cai=0.00007;
    const Real Nai=7.67;
    const Real Ki=138.3;

//Constants
    const Real R=8314.472;
    const Real F=96485.3415f;
    const Real T=310.0;
    const Real RTONF=(R*T)/F;

//Parameters for currents
//Parameters for IKr
    const Real Gkr=0.101;
//Parameters for Iks
    const Real pKNa=0.03;
#ifdef EPI
    const Real Gks=0.257;
#endif
#ifdef ENDO
    const Real Gks=0.392;
#endif
#ifdef MCELL
    const Real Gks=0.098;
#endif
//Parameters for Ik1
    const Real GK1=5.405;
//Parameters for Ito
#ifdef EPI
    const Real Gto=0.294;
#endif
#ifdef ENDO
    const Real Gto=0.073;
#endif
#ifdef MCELL
    const Real Gto=0.294;
#endif
//Parameters for INa
    const Real GNa=14.838;
//Parameters for IbNa
    const Real GbNa=0.00029;
//Parameters for INaK
    const Real KmK=1.0;
    const Real KmNa=40.0;
    const Real knak=2.724;
//Parameters for ICaL
    const Real GCaL=0.2786f*pcal;
//Parameters for IbCa
    const Real GbCa=0.000592;
//Parameters for INaCa
    const Real knaca=1000;
    const Real KmNai=87.5;
    const Real KmCa=1.38;
    const Real ksat=0.1;
    const Real n=0.35;
//Parameters for IpCa
    const Real GpCa=0.1238;
    const Real KpCa=0.0005;
//Parameters for IpK;
    const Real GpK=0.0293;


    const Real Ek=RTONF*(logf((Ko/Ki)));
    const Real Ena=RTONF*(logf((Nao/Nai)));
    const Real Eks=RTONF*(logf((Ko+pKNa*Nao)/(Ki+pKNa*Nai)));
    const Real Eca=0.5f*RTONF*(logf((Cao/Cai)));
    Real IKr;
    Real IKs;
    Real IK1;
    Real Ito;
    Real INa;
    Real IbNa;
    Real ICaL;
    Real IbCa;
    Real INaCa;
    Real IpCa;
    Real IpK;
    Real INaK;
    Real IKatp;

    Real Ak1;
    Real Bk1;
    Real rec_iK1;
    Real rec_ipK;
    Real rec_iNaK;
    Real AM;
    Real BM;
    Real AH_1;
    Real BH_1;
    Real AH_2;
    Real BH_2;
    Real AJ_1;
    Real BJ_1;
    Real AJ_2;
    Real BJ_2;
    Real M_INF;
    Real H_INF;
    Real J_INF;
    Real TAU_M;
    Real TAU_H;
    Real TAU_J;
    Real axr1;
    Real bxr1;
    Real Xr1_INF;
    Real Xr2_INF_new;
    Real TAU_Xr1;
    Real Axs;
    Real Bxs;
    Real Xs_INF;
    Real TAU_Xs;
    Real R_INF_new;
    Real S_INF;
    Real TAU_S;
    Real Af;
    Real Bf;
    Real Cf;
    Real Af2;
    Real Bf2;
    Real Cf2;
    Real D_INF_new;
    Real TAU_F;
    Real F_INF;
    Real TAU_F2;
    Real F2_INF;
    Real sItot;


    //Needed to compute currents
    Ak1=0.1f/(1.0f+expf(0.06f*(svolt-Ek-200.0f)));
    Bk1=(3.0f*expf(0.0002f*(svolt-Ek+100.0f))+
         expf(0.1f*(svolt-Ek-10.0f)))/(1.0f+expf(-0.5f*(svolt-Ek)));
    rec_iK1=Ak1/(Ak1+Bk1);
    rec_iNaK=(1.0f/(1.0f+0.1245f*expf(-0.1f*svolt*F/(R*T))+0.0353f*expf(-svolt*F/(R*T))));
    rec_ipK=1.0f/(1.0f+expf((25.0f-svolt)/5.98f));


    //Compute currents
    INa=GNa*sm*sm*sm*sh*sj*(svolt-Ena);
    ICaL=GCaL*D_INF*sf*sf2*(svolt-60);
    Ito=Gto*R_INF*ss*(svolt-Ek);
    IKr=Gkr*sqrtf(Ko/5.4f)*sxr1*Xr2_INF*(svolt-Ek);
    IKs=Gks*sxs*sxs*(svolt-Eks);
    IK1=GK1*rec_iK1*(svolt-Ek);
    INaCa=knaca*(1.0f/(KmNai*KmNai*KmNai+Nao*Nao*Nao))*(1.0f/(KmCa+Cao))*
          (1.0f/(1.0f+ksat*expf((n-1.0f)*svolt*F/(R*T))))*
          (expf(n*svolt*F/(R*T))*Nai*Nai*Nai*Cao-
           expf((n-1.0f)*svolt*F/(R*T))*Nao*Nao*Nao*Cai*2.5f);
    INaK=knak*(Ko/(Ko+KmK))*(Nai/(Nai+KmNa))*rec_iNaK;
    IpCa=GpCa*Cai/(KpCa+Cai);
    IpK=GpK*rec_ipK*(svolt-Ek);
    IbNa=GbNa*(svolt-Ena);
    IbCa=GbCa*(svolt-Eca);

    IKatp = gkbaratp*(svolt-Ek);

    //Determine total current
    (sItot) = IKr    +
              IKs   +
              IK1   +
              Ito   +
              INa   +
              IbNa  +
              ICaL  +
              IbCa  +
              INaK  +
              INaCa +
              IpCa  +
              IpK   +
              IKatp +
              stim_current;

    //compute steady state values and time constants
    AM=1.0f/(1.0f+expf((-60.0f-svolt)/5.0f));
    BM=0.1f/(1.0f+expf((svolt+35.0f)/5.0f))+0.10f/(1.0f+expf((svolt-50.0f)/200.0f));
    TAU_M=AM*BM;
    M_INF=1.0f/((1.0f+expf((-56.86f-svolt)/9.03f))*(1.0f+expf((-56.86f-svolt)/9.03f)));
    if (svolt>=-40.)
    {
        AH_1=0.0f;
        BH_1=(0.77f/(0.13f*(1.0f+expf(-(svolt+10.66f)/11.1f))));
        TAU_H= 1.0f/(AH_1+BH_1);
    }
    else
    {
        AH_2=(0.057f*expf(-(svolt+80.0f)/6.8f));
        BH_2=(2.7f*expf(0.079f*svolt)+(3.1e5f)*expf(0.3485f*svolt));
        TAU_H=1.0f/(AH_2+BH_2);
    }
    H_INF=1.0f/((1.0f+expf((svolt+71.55f)/7.43f))*(1.0f+expf((svolt+71.55f)/7.43f)));
    if(svolt>=-40.0f)
    {
        AJ_1=0.0f;
        BJ_1=(0.6f*expf((0.057f)*svolt)/(1.0f+expf(-0.1f*(svolt+32.0f))));
        TAU_J= 1.0f/(AJ_1+BJ_1);
    }
    else
    {
        AJ_2=(((-2.5428e4f)*expf(0.2444f*svolt)-(6.948e-6f)*expf(-0.04391f*svolt))*(svolt+37.78f)/
              (1.0f+expf(0.311f*(svolt+79.23f))));
        BJ_2=(0.02424f*expf(-0.01052f*svolt)/(1.0f+expf(-0.1378f*(svolt+40.14f))));
        TAU_J= 1.0f/(AJ_2+BJ_2);
    }
    J_INF=H_INF;

    Xr1_INF=1.0f/(1.0f+expf((-26.0f-svolt)/7.0f));
    axr1=450.0f/(1.0f+expf((-45.0f-svolt)/10.0f));
    bxr1=6.0f/(1.0f+expf((svolt-(-30.0f))/11.5f));
    TAU_Xr1=axr1*bxr1;
    Xr2_INF_new=1.0f/(1.0f+expf((svolt-(-88.0f))/24.0f));


    Xs_INF=1.0f/(1.0f+expf((-5.0f-svolt)/14.0f));
    Axs=(1400.0f/(sqrtf(1.0f+expf((5.0f-svolt)/6.0f))));
    Bxs=(1.0f/(1.0f+expf((svolt-35.0f)/15.0f)));
    TAU_Xs=Axs*Bxs+80;

#ifdef EPI
    R_INF_new=1./(1.+expf((20-svolt)/6.));
    S_INF=1./(1.+expf((svolt+20)/5.));
    TAU_S=85.*expf(-(svolt+45.)*(svolt+45.)/320.)+5./(1.+expf((svolt-20.)/5.))+3.;
#endif
#ifdef ENDO
    R_INF_new=1.0f/(1.0f+expf((20.0f-svolt)/6.0f));
    S_INF=1.0f/(1.0f+expf((svolt+28.0f)/5.0f));
    TAU_S=1000.0f*expf(-(svolt+67.0f)*(svolt+67.0f)/1000.0f)+8.0f;
#endif
#ifdef MCELL
    R_INF_new=1./(1.+expf((20-svolt)/6.));
    S_INF=1./(1.+expf((svolt+20)/5.));
    TAU_S=85.*expf(-(svolt+45.)*(svolt+45.)/320.)+5./(1.+expf((svolt-20.)/5.))+3.;
#endif


    D_INF_new=1.0f/(1.0f+expf((-8.0f-svolt)/7.5f));
    F_INF=1.0f/(1.0f+expf((svolt+20)/7));
    Af=1102.5f*expf(-(svolt+27)*(svolt+27.0f)/225.0f);
    Bf=200.0f/(1.0f+expf((13.0f-svolt)/10.f));
    Cf=(180.0f/(1.0f+expf((svolt+30.0f)/10.0f)))+20.0f;
    TAU_F=Af+Bf+Cf;
    F2_INF=0.67f/(1.0f+expf((svolt+35.0f)/7.0f))+0.33f;
    Af2=600.0f*expf(-(svolt+27.0f)*(svolt+27.0f)/170.0f);
    Bf2=7.75f/(1.0f+expf((25.0f-svolt)/10.0f));
    Cf2=16.0f/(1.0f+expf((svolt+30.0f)/10.0f));
    TAU_F2=Af2+Bf2+Cf2;

    //update voltage
    rDY_[0] = -sItot;

    //Update gates
    rDY_[1] = M_INF-(M_INF-sm)*expf(-dt/TAU_M);
    rDY_[2] = H_INF-(H_INF-sh)*expf(-dt/TAU_H);
    rDY_[3] = J_INF-(J_INF-sj)*expf(-dt/TAU_J);
    rDY_[4] = Xr1_INF-(Xr1_INF-sxr1)*expf(-dt/TAU_Xr1);
    rDY_[5] = Xs_INF-(Xs_INF-sxs)*expf(-dt/TAU_Xs);
    rDY_[6]= S_INF-(S_INF-ss)*expf(-dt/TAU_S);
    rDY_[7] =F_INF-(F_INF-sf)*expf(-dt/TAU_F);
    rDY_[8] =F2_INF-(F2_INF-sf2)*expf(-dt/TAU_F2);

    rDY_[9] = D_INF_new;
    rDY_[10] = R_INF_new;
    rDY_[11] = Xr2_INF_new;


}
